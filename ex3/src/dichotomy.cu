#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "dichotomy.h"

bool f(double x, double y, double *error){
    if( (-x*x + eps_r * y*y) < 0 ){
        return false;
    }

    if( (x*x - y*y) == 0 ){
        return false;
    }

    if( (-x*x + eps_r * y*y)/(x*x - y*y) < 0 ){
        return false;
    }

    *error = tan( sqrt( -x*x + eps_r * y*y ) ) - sqrt( ((-x*x + eps_r * y*y)/(x*x - y*y)) );
    return true;
}

bool dichotomy(double x, double min_y, double max_y, double error, double *ans){
    const int max_trials = 1e4;
    
    double t0, t1;
    f(x, min_y, &t0);
    f(x, max_y, &t1);

    if( t0 * t1 > 0 ){
        return false;
    }

    int num = 0;
    double mid_y = (max_y + min_y) / 2.0;
    f(x, mid_y, &t1);
    while (abs(t1) > error) {
        f(x, min_y, &t0);
        f(x, mid_y, &t1);

        if( t0 * t1 >= 0 ){
            min_y = mid_y;
        }else {
            max_y = mid_y;
        }

        mid_y = (max_y + min_y) / 2.0;

        if(num > max_trials){
            return false;
        }

        num ++;
    }

    f(x, mid_y, ans);
    return true;
}